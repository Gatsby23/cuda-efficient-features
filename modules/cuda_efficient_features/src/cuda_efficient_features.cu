#include "hip/hip_runtime.h"
/*
Copyright 2023 Fixstars Corporation

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

http ://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
*/

#include "cuda_efficient_features.h"

#include <opencv2/core/cuda.hpp>

#include <hip/hip_runtime.h>
#include <>
#include <thrust/device_ptr.h>
#include <thrust/scan.h>
#include <thrust/sort.h>
#include <thrust/execution_policy.h>

#include "cuda_macro.h"

namespace cv
{
namespace cuda
{

static constexpr int CELL_SIZE = 16;
static constexpr int PATCH_SIZE = 31;
static constexpr int HALF_PATCH_SIZE = 15;
static constexpr float HARRIS_K = 0.04f;

static constexpr int LOCATION_ROW = EfficientFeatures::LOCATION_ROW;
static constexpr int RESPONSE_ROW = EfficientFeatures::RESPONSE_ROW;
static constexpr int ANGLE_ROW    = EfficientFeatures::ANGLE_ROW;
static constexpr int OCTAVE_ROW   = EfficientFeatures::OCTAVE_ROW;
static constexpr int SIZE_ROW     = EfficientFeatures::SIZE_ROW;
static constexpr int ROWS_COUNT   = EfficientFeatures::ROWS_COUNT;

static __device__ inline int distanceSq(short2 pt1, short2 pt2)
{
	const int dx = pt1.x - pt2.x;
	const int dy = pt1.y - pt2.y;
	return dx * dx + dy * dy;
}

static __device__ inline float convertToDegree(float angle)
{
	constexpr float PI = static_cast<float>(CV_PI);
	if (angle < 0)
		angle += 2.f * PI;
	return (180.f / PI) * angle;
}

static __device__ inline bool IsMaxPoint(int idx1, const short2* points, const float* responses,
	const int* blockPtr, const int* pointIds, int gridW, int gridH, int imageRadius, int blockRadius)
{
	const short2 pt1 = points[idx1];
	const float response1 = responses[idx1];

	const int bx1 = pt1.x / CELL_SIZE;
	const int by1 = pt1.y / CELL_SIZE;

	const int minx = ::max(bx1 - blockRadius, 0);
	const int maxx = ::min(bx1 + blockRadius, gridW - 1);
	const int miny = ::max(by1 - blockRadius, 0);
	const int maxy = ::min(by1 + blockRadius, gridH - 1);

	for (int by = miny; by <= maxy; by++)
	{
		for (int bx = minx; bx <= maxx; bx++)
		{
			const int blockId = by * gridW + bx;
			for (int k = blockPtr[blockId]; k < blockPtr[blockId + 1]; k++)
			{
				const int idx2 = pointIds[k];
				if (idx1 == idx2)
					continue;

				const short2 pt2 = points[idx2];
				const float response2 = responses[idx2];

				if (response1 <= response2 && distanceSq(pt1, pt2) < imageRadius)
					return false;
			}
		}
	}

	return true;
};

static __device__ float calcResponse(PtrStepb image, short2 pt)
{
	constexpr int BLOCK_SIZE = 7;
	constexpr int RADIUS = BLOCK_SIZE / 2;
	constexpr float SCALE = 1.f / (4 * BLOCK_SIZE * 255);

	const int x0 = pt.x;
	const int y0 = pt.y;

	float sxx = 0, sxy = 0, syy = 0;
	for (int iy = -RADIUS; iy <= RADIUS; ++iy)
	{
		for (int ix = -RADIUS; ix <= RADIUS; ++ix)
		{
			const int x = x0 + ix;
			const int y = y0 + iy;

			const int v00 = image(y - 1, x - 1);
			const int v01 = image(y - 1, x);
			const int v02 = image(y - 1, x + 1);

			const int v10 = image(y, x - 1);
			const int v12 = image(y, x + 1);

			const int v20 = image(y + 1, x - 1);
			const int v21 = image(y + 1, x);
			const int v22 = image(y + 1, x + 1);

			const float dx = SCALE * ((v02 + 2 * v12 + v22) - (v00 + 2 * v10 + v20));
			const float dy = SCALE * ((v20 + 2 * v21 + v22) - (v00 + 2 * v01 + v02));
			sxx += dx * dx;
			sxy += dx * dy;
			syy += dy * dy;
		}
	}

	const float detM = sxx * syy - sxy * sxy;
	const float trM = sxx + syy;

	return detM - HARRIS_K * trM * trM;
}

static __device__ float IC_Angle(PtrStepb image, short2 pt)
{
	constexpr int U_MAX[] = { 15, 15, 15, 15, 14, 14, 14, 13, 13, 12, 11, 10, 9, 8, 6, 3, 0 };

	const int x = pt.x;
	const int y = pt.y;

	int m_01 = 0, m_10 = 0;

	// Treat the center line differently, v=0
	for (int dx = -HALF_PATCH_SIZE; dx <= HALF_PATCH_SIZE; ++dx)
		m_10 += dx * image(y, x + dx);

	// Go line by line in the circuI853lar patch
	for (int dy = 1; dy <= HALF_PATCH_SIZE; ++dy)
	{
		// Proceed over the two lines
		int y_sum = 0;
		const int d = U_MAX[dy];
		for (int dx = -d; dx <= d; ++dx)
		{
			const int valT = image(y - dy, x + dx);
			const int valB = image(y + dy, x + dx);

			y_sum += (valB - valT);
			m_10 += dx * (valB + valT);
		}
		m_01 += dy * y_sum;
	}

	return convertToDegree(::atan2f((float)m_01, (float)m_10));
}

__global__ void nptPerBlockKernel(const short2* points, int npoints, int* nptPerBlock, int gridStep)
{
	const int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= npoints)
		return;

	const short2 pt = points[i];
	const int bx = pt.x / CELL_SIZE;
	const int by = pt.y / CELL_SIZE;
	const int blockId = by * gridStep + bx;
	atomicAdd(&nptPerBlock[blockId], 1);
}

__global__ void assignIndexKernel(const short2* points, int npoints, int* pointIds, int* nptPerBlock, int gridStep)
{
	const int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= npoints)
		return;

	const short2 pt = points[i];
	const int cx = pt.x / CELL_SIZE;
	const int cy = pt.y / CELL_SIZE;
	const int blockId = cy * gridStep + cx;

	const int k = atomicAdd(&nptPerBlock[blockId], 1);
	pointIds[k] = i;
}

__global__ void radiusSuppressionKernel(const short2* srcPts, const float* srcRes, int npoints,
	short2* dstPts, float* dstRes, int* count, const int* blockPtr, const int* pointIds,
	int gridW, int gridH, int imageRadius, int blockRadius)
{
	const int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= npoints)
		return;

	if (IsMaxPoint(i, srcPts, srcRes, blockPtr, pointIds, gridW, gridH, imageRadius, blockRadius))
	{
		const int k = atomicAdd(count, 1);
		dstPts[k] = srcPts[i];
		dstRes[k] = srcRes[i];
	}
}

__global__ void calcResponsesKernel(PtrStepb image, const short2* points, float* responses, int npoints)
{
	const int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= npoints)
		return;

	responses[i] = calcResponse(image, points[i]);
}

__global__ void calcAnglesKernel(PtrStepb image, const short2* points, float* angles, int npoints)
{
	const int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= npoints)
		return;

	angles[i] = IC_Angle(image, points[i]);
}

__global__ void scalePointsKernel(short2* points, int* octaves, float* sizes, int npoints, float scale, int octave)
{
	const int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= npoints)
		return;

	short2 pt = points[i];
	pt.x = static_cast<short>(scale * pt.x + 0.5f);
	pt.y = static_cast<short>(scale * pt.y + 0.5f);
	points[i] = pt;
	octaves[i] = octave;
	sizes[i] = scale * PATCH_SIZE;
}

__global__ void convertKeypointsKernel(const short2* srcLoc, const float* srcAngles, float4* dstKeypoints, int npoints)
{
	const int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= npoints)
		return;

	const short2 pt = srcLoc[i];
	float4 kpt;
	kpt.x = pt.x;
	kpt.y = pt.y;
	kpt.z = PATCH_SIZE;
	kpt.w = srcAngles[i];
	dstKeypoints[i] = kpt;
}

static void exclusiveScan(const int* src, int* dst, int size, hipStream_t stream = 0)
{
	auto ptrSrc = thrust::device_pointer_cast(src);
	auto ptrDst = thrust::device_pointer_cast(dst);
	thrust::exclusive_scan(thrust::cuda::par.on(stream), ptrSrc, ptrSrc + size, ptrDst);
}

int radiusSuppressionBufferSize(Size imgSize, int npoints)
{
	const int gridW = divUp(imgSize.width, CELL_SIZE);
	const int gridH = divUp(imgSize.height, CELL_SIZE);
	const int nblocks = gridW * gridH;
	const int ptrSize = nblocks + 1;
	return 2 * ptrSize + npoints;
}

void radiusSuppression(const GpuMat& src, GpuMat& dst, Size imgSize, float radius,
	GpuMat& d_buffer, HostMem& h_buffer, hipStream_t stream)
{
	const int npoints = src.cols;
	if (npoints <= 0)
		return;

	const int imageRadius = cvCeil(radius * radius);
	const int blockRadius = cvCeil(radius / CELL_SIZE);

	struct Config { int block, grid; } cfg;
	cfg.block = 512;
	cfg.grid = divUp(npoints, cfg.block);

	const int gridW = divUp(imgSize.width, CELL_SIZE);
	const int gridH = divUp(imgSize.height, CELL_SIZE);
	const int nblocks = gridW * gridH;
	const int ptrSize = nblocks + 1;

	CV_Assert(dst.rows >= ROWS_COUNT && dst.cols >= npoints && dst.type() == CV_32F);
	CV_Assert(d_buffer.size().area() >= 2 * ptrSize + npoints);
	CV_Assert(h_buffer.size().area() >= 1);

	const short2* srcPts = src.ptr<short2>(LOCATION_ROW);
	const float* srcRes = src.ptr<float>(RESPONSE_ROW);
	short2* dstPts = dst.ptr<short2>(LOCATION_ROW);
	float* dstRes = dst.ptr<float>(RESPONSE_ROW);

	int* nptPerBlock = d_buffer.ptr<int>();
	int* blockPtr = d_buffer.ptr<int>() + 1 * ptrSize;
	int* pointIds = d_buffer.ptr<int>() + 2 * ptrSize;
	int* d_count = nptPerBlock + nblocks;
	int* h_count = h_buffer.createMatHeader().ptr<int>();

	// count number of points per block
	CUDA_CHECK(hipMemsetAsync(nptPerBlock, 0, sizeof(int) * ptrSize, stream));
	nptPerBlockKernel<<<cfg.grid, cfg.block, 0, stream>>>(srcPts, npoints, nptPerBlock, gridW);
	CUDA_CHECK(hipGetLastError());

	// calculate start addresses corresponding to each blocks
	exclusiveScan(nptPerBlock, blockPtr, ptrSize, stream);

	// assign point indices to blocks
	CUDA_CHECK(hipMemcpyAsync(nptPerBlock, blockPtr, sizeof(int) * nblocks, hipMemcpyDeviceToDevice, stream));
	assignIndexKernel<<<cfg.grid, cfg.block, 0, stream>>>(srcPts, npoints, pointIds, nptPerBlock, gridW);
	CUDA_CHECK(hipGetLastError());

	// radius suppression
	radiusSuppressionKernel<<<cfg.grid, cfg.block, 0, stream>>>(srcPts, srcRes, npoints, dstPts, dstRes, d_count,
		blockPtr, pointIds, gridW, gridH, imageRadius, blockRadius);
	CUDA_CHECK(hipGetLastError());

	// get number of remaining points 
	CUDA_CHECK(hipMemcpyAsync(h_count, d_count, sizeof(int), hipMemcpyDeviceToHost, stream));

	CUDA_CHECK(hipStreamSynchronize(stream));

	dst.cols = *h_count;
}

void limitPoints(GpuMat& points, int maxpoints, hipStream_t stream)
{
	const int npoints = points.cols;
	if (npoints <= maxpoints)
		return;

	auto locations = thrust::device_pointer_cast(points.ptr<short2>(0));
	auto responses = thrust::device_pointer_cast(points.ptr<float>(1));

	thrust::sort_by_key(thrust::cuda::par.on(stream), responses, responses + npoints, locations, thrust::greater<float>());

	points.cols = maxpoints;

	CUDA_CHECK(hipGetLastError());
}

void calcResponses(const GpuMat& image, GpuMat& points, hipStream_t stream)
{
	const int npoints = points.cols;
	if (npoints <= 0)
		return;

	const int block = 512;
	const int grid = divUp(npoints, block);

	const short2* locations = points.ptr<short2>(LOCATION_ROW);
	float* responses = points.ptr<float>(RESPONSE_ROW);

	calcResponsesKernel<<<grid, block, 0, stream>>>(image, locations, responses, npoints);
	CUDA_CHECK(hipGetLastError());
}

void calcAngles(const GpuMat& image, GpuMat& points, hipStream_t stream)
{
	const int npoints = points.cols;
	if (npoints <= 0)
		return;

	const int block = 512;
	const int grid = divUp(npoints, block);

	const short2* locations = points.ptr<short2>(LOCATION_ROW);
	float* angles = points.ptr<float>(ANGLE_ROW);

	calcAnglesKernel<<<grid, block, 0, stream>>>(image, locations, angles, npoints);
	CUDA_CHECK(hipGetLastError());
}

void scalePoints(GpuMat& points, float scale, int octave, hipStream_t stream)
{
	const int npoints = points.cols;
	if (npoints <= 0)
		return;

	const int block = 512;
	const int grid = divUp(npoints, block);

	short2* locations = points.ptr<short2>(LOCATION_ROW);
	int* octaves = points.ptr<int>(OCTAVE_ROW);
	float* sizes = points.ptr<float>(SIZE_ROW);

	scalePointsKernel<<<grid, block, 0, stream>>>(locations, octaves, sizes, npoints, scale, octave);
	CUDA_CHECK(hipGetLastError());
}

void convertKeypoints(const GpuMat& src, GpuMat& dst, hipStream_t stream)
{
	const int npoints = src.cols;
	if (npoints <= 0)
	{
		dst.release();
		return;
	}

	const int block = 512;
	const int grid = divUp(npoints, block);

	CV_Assert(dst.rows >= npoints && dst.cols >= 1 && dst.type() == CV_32FC4);

	const short2* srcLoc = src.ptr<short2>(LOCATION_ROW);
	const float* srcAngles = src.ptr<float>(ANGLE_ROW);
	float4* dstKeypoints = dst.ptr<float4>(0);

	convertKeypointsKernel<<<grid, block, 0, stream>>>(srcLoc, srcAngles, dstKeypoints, npoints);
	CUDA_CHECK(hipGetLastError());
}

} // namespace cuda
} // namespace cv
